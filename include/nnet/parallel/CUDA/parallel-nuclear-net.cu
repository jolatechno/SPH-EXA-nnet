#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Definition of CUDA integration functions.
 *
 * @author Joseph Touzet <joseph.touzet@ens-paris-saclay.fr>
 */


#include "parallel-nuclear-net.cuh"

namespace nnet::parallel_nnet {
	/***********************************************/
	/* code to compute nuclear reaction on the GPU */
	/***********************************************/


	/*! @brief kernel that integrate nuclear reaction over a given timestep in parallel on device
	 * 
	 * called in cudaComputeNuclearReactions, should not be directly accessed by user
	 */
	template<class func_type, class func_eos, typename Float>
	__global__ void cudaKernelComputeNuclearReactions(const size_t n_particles, const int dimension,
		Float *global_buffer,
		Float *rho_, Float *rho_m1_, Float **Y_, Float *temp_, Float *dt_,
		const Float hydro_dt, const Float previous_dt,
		const nnet::gpu_reaction_list *reactions, const func_type *construct_rates_BE, const func_eos *eos,
		bool use_drhodt)
	{
	    size_t thread = blockIdx.x*blockDim.x + threadIdx.x;
	    if (thread < n_particles) {
	    	// buffer sizes
		    const size_t Y_size        =  dimension;
		    const size_t Mp_size       = (dimension + 1)*(dimension + 1);
		    const size_t RHS_size      =  dimension + 1;
		    const size_t DY_T_size     =  dimension + 1;
		    const size_t Y_buffer_size =  dimension;
		    const size_t rates_size    =  reactions->size();

		    // allocate local buffer
		    Float T_buffer;
		    Float *Y         = global_buffer + (Y_size + Mp_size + RHS_size + DY_T_size + Y_buffer_size + rates_size)*(blockIdx.x*blockDim.x + threadIdx.x);
	    	Float *Mp        = Y        + Y_size;
			Float *RHS       = Mp       + Mp_size;
			Float *DY_T      = RHS      + RHS_size;
			Float *Y_buffer  = DY_T     + DY_T_size;
			Float *rates     = Y_buffer + Y_buffer_size;

	    	 // copy Y to local buffer
			for (int j = 0; j < dimension; ++j)
				Y[j] = Y_[j][thread];

			// compute drho/dt
			Float drho_dt = 0;
			if (use_drhodt && rho_m1_[thread] != 0)
				drho_dt = (rho_[thread] - rho_m1_[thread])/previous_dt;

			// initial condition
			Float elapsed = 0.0;
			// run simulation
			for (int iter = 1;; ++iter) {
				// generate system
				nnet::prepare_system_substep(dimension,
					Mp, RHS, rates,
					*reactions, *construct_rates_BE, *eos,
					Y, temp_[thread], Y_buffer, T_buffer,
					rho_[thread], drho_dt,
					hydro_dt, elapsed, dt_[thread], iter);

				// solve M*D{T, Y} = RHS
				eigen::solve(Mp, RHS, DY_T, dimension + 1, (Float)nnet::constants::epsilon_system);

				// finalize
				if(nnet::finalize_system_substep(dimension,
					Y, temp_[thread],
					Y_buffer, T_buffer,
					DY_T, hydro_dt, elapsed,
					dt_[thread], iter))
				{
					// copy Y "buffer" back to actual storage
					for (int j = 0; j < dimension; ++j)
						Y_[j][thread] = Y[j];

					// exit
					break;
				}
			}
	    }
	}


	/*! @brief function that integrate nuclear reaction over a given timestep in parallel on device
	 * 
	 * used in include/nnet/sphexa/nuclear-net.hpp, should not be directly accessed by user
	 */
	template<class func_type, class func_eos, typename Float>
	void cudaComputeNuclearReactions(const size_t n_particles, const int dimension,
		thrust::device_vector<Float> &buffer,
		Float *rho_, Float *rho_m1_, Float **Y_, Float *temp_, Float *dt_,
		const Float hydro_dt, const Float previous_dt,
		const nnet::gpu_reaction_list &reactions, const func_type &construct_rates_BE, const func_eos &eos,
		bool use_drhodt)
	{
		// copy classes to gpu
		nnet::gpu_reaction_list *dev_reactions;
		func_type               *dev_construct_rates_BE;
		func_eos                *dev_eos;
		// allocate
		gpuErrchk(hipMalloc((void**)&dev_reactions,          sizeof(nnet::gpu_reaction_list)));
		gpuErrchk(hipMalloc((void**)&dev_construct_rates_BE, sizeof(func_type)));
		gpuErrchk(hipMalloc((void**)&dev_eos,                sizeof(func_eos)));
		// actually copy
		gpuErrchk(hipMemcpy(dev_reactions,          &reactions,          sizeof(nnet::gpu_reaction_list), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(dev_construct_rates_BE, &construct_rates_BE, sizeof(func_type),               hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(dev_eos,                &eos,                sizeof(func_eos),                hipMemcpyHostToDevice));

		// compute chunk sizes
		int cuda_num_blocks = (n_particles + constants::cuda_num_thread_per_block_nnet - 1)/constants::cuda_num_thread_per_block_nnet;

		// buffer sizes
		const size_t Y_size        =  dimension;
	    const size_t Mp_size       = (dimension + 1)*(dimension + 1);
	    const size_t RHS_size      =  dimension + 1;
	    const size_t DY_T_size     =  dimension + 1;
	    const size_t Y_buffer_size =  dimension;
	    const size_t rates_size    =  reactions.size();
		// allocate global buffer
	    const size_t buffer_size = (Y_size + Mp_size + RHS_size + DY_T_size + Y_buffer_size + rates_size)*n_particles;
		if (buffer.size() < buffer_size)
			buffer.resize(buffer_size);


		// launch kernel
	    cudaKernelComputeNuclearReactions<<<cuda_num_blocks, constants::cuda_num_thread_per_block_nnet>>>(n_particles, dimension,
	(Float*)thrust::raw_pointer_cast(buffer.data()),
			rho_, rho_m1_, Y_, temp_, dt_,
			hydro_dt, previous_dt,
			dev_reactions, dev_construct_rates_BE, dev_eos,
			use_drhodt);


		// debuging: check for error
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

	    // free cuda classes
	    gpuErrchk(hipFree(dev_reactions));
	    gpuErrchk(hipFree(dev_construct_rates_BE));
	    gpuErrchk(hipFree(dev_eos));
	}


	// used templates:
	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<double>&, double*, double*, double**, double*, double*, const double, const double,
		nnet::gpu_reaction_list const&, nnet::net87::compute_reaction_rates_functor const&, nnet::eos::ideal_gas_functor const&,
		bool);
	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<double>&, double*, double*, double**, double*, double*, const double, const double,
		nnet::gpu_reaction_list const&, nnet::net86::compute_reaction_rates_functor const&, nnet::eos::ideal_gas_functor const&,
		bool);
	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<double>&, double*, double*, double**, double*, double*, const double, const double,
		nnet::gpu_reaction_list const&, nnet::net14::compute_reaction_rates_functor const&, nnet::eos::ideal_gas_functor const&,
		bool);

	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<double>&, double*, double*, double**, double*, double*, const double, const double,
		nnet::gpu_reaction_list const&, nnet::net87::compute_reaction_rates_functor const&, nnet::eos::helmholtz_functor<double> const&,
		bool);
	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<double>&, double*, double*, double**, double*, double*, const double, const double,
		nnet::gpu_reaction_list const&, nnet::net86::compute_reaction_rates_functor const&, nnet::eos::helmholtz_functor<double> const&,
		bool);
	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<double>&, double*, double*, double**, double*, double*, const double, const double,
		nnet::gpu_reaction_list const&, nnet::net14::compute_reaction_rates_functor const&, nnet::eos::helmholtz_functor<double> const&,
		bool);

	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<float>&, float*, float*, float**, float*, float*, const float, const float,
		nnet::gpu_reaction_list const&, nnet::net87::compute_reaction_rates_functor const&, nnet::eos::ideal_gas_functor const&,
		bool);
	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<float>&, float*, float*, float**, float*, float*, const float, const float,
		nnet::gpu_reaction_list const&, nnet::net86::compute_reaction_rates_functor const&, nnet::eos::ideal_gas_functor const&,
		bool);
	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<float>&, float*, float*, float**, float*, float*, const float, const float,
		nnet::gpu_reaction_list const&, nnet::net14::compute_reaction_rates_functor const&, nnet::eos::ideal_gas_functor const&,
		bool);

	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<float>&, float*, float*, float**, float*, float*, const float, const float,
		nnet::gpu_reaction_list const&, nnet::net87::compute_reaction_rates_functor const&, nnet::eos::helmholtz_functor<float> const&,
		bool);
	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<float>&, float*, float*, float**, float*, float*, const float, const float,
		nnet::gpu_reaction_list const&, nnet::net86::compute_reaction_rates_functor const&, nnet::eos::helmholtz_functor<float> const&,
		bool);
	template void cudaComputeNuclearReactions(const unsigned long, const int,
		thrust::device_vector<float>&, float*, float*, float**, float*, float*, const float, const float,
		nnet::gpu_reaction_list const&, nnet::net14::compute_reaction_rates_functor const&, nnet::eos::helmholtz_functor<float> const&,
		bool);



	/************************************************************/
	/* code to compute helmholtz equation of a state on the GPU */
	/************************************************************/


	/*! @brief kernel that computes helmholtz EOS in parallel on device
	 * 
	 * called in cudaComputeHelmholtz, should not be directly accessed by user
	 */
	template<typename Float /*, class func_eos*/>
	__global__ void cudaKernelComputeHelmholtz(const size_t n_particles, const int dimension, const Float *Z,
		const Float *temp_, const Float *rho_, Float *const* Y_, 
		Float *u, Float *cv, Float *p, Float *c, Float *dpdT)
	{
		size_t thread = blockIdx.x*blockDim.x + threadIdx.x;
		if (thread < n_particles) {
			// compute abar and zbar
			double abar = 0, zbar = 0;
			for (int i = 0; i < dimension; ++i) {
				abar += Y_[i][thread];
				zbar += Y_[i][thread]*Z[i];
			}


			// actually compute helmholtz eos
			auto eos_struct = nnet::eos::helmholtz(abar, zbar, temp_[thread], rho_[thread]);


			// copy results to buffers
			u[thread]    = eos_struct.u;
			cv[thread]   = eos_struct.cv;
			p[thread]    = eos_struct.p;
			c[thread]    = eos_struct.c;
			dpdT[thread] = eos_struct.dpdT;
		}
	}


	/*! @brief function that computes helmholtz EOS in parallel on device
	 * 
	 * used in include/nnet/sphexa/nuclear-net.hpp, should not be directly accessed by user
	 */
	template<typename Float>
	void cudaComputeHelmholtz(const size_t n_particles, const int dimension, const Float *Z,
		const Float *temp_, const Float *rho_, Float *const* Y_,
		Float *u, Float *cv, Float *p, Float *c, Float *dpdT)
	{
		// compute chunk sizes
		int cuda_num_blocks = (n_particles + constants::cuda_num_thread_per_block - 1)/constants::cuda_num_thread_per_block;
		
		// launch kernel
	    cudaKernelComputeHelmholtz<<<cuda_num_blocks, constants::cuda_num_thread_per_block>>>(n_particles, dimension, Z,
			temp_, rho_, Y_,
			u, cv, p, c, dpdT);
	}


	// used templates:
	template void cudaComputeHelmholtz(const size_t n_particles, const int dimension, const double *Z,
		const double *temp_, const double *rho_, double *const* Y_,
		double *u, double *cv, double *p, double *c, double *dpdT);
	template void cudaComputeHelmholtz(const size_t n_particles, const int dimension, const float *Z,
		const float *temp_, const float *rho_, float *const* Y_,
		float *u, float *cv, float *p, float *c, float *dpdT);
}