#include "hip/hip_runtime.h"
#pragma once

#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "nuclear-net.cuh"

#ifndef CUDA_BLOCK_SIZE
	#define CUDA_BLOCK_SIZE 256
#endif

namespace nnet {
namespace constants {
	extern __device__ double dev_min_rho, dev_min_temp;
}

extern "C" {
	// forward definition
	// template<class func_type /*, class func_eos*/ /*, typename Float*/>
	__host__ __device__ void solve_system_substep(const int dimension,
		Float *Mp, Float *RHS, Float *DY_T, Float *rates, Float *drates_dT,
		const ptr_reaction_list &reactions, const func_type &construct_rates_BE, const func_eos &eos,
		Float *final_Y, Float &final_T, Float *Y_buffer,
		const Float final_rho, const Float drho_dt, Float const dt_tot, Float &dt);
}
}

namespace sphexa {
namespace sphnnet {
	// template<class func_type /*, class func_eos*/ /*, typename Float*/>
	__global__ void cudaKernelComputeNuclearReactions(const int n_particles, const int dimension,
	Float *rho_, Float *previous_rho_, Float *Y_, Float *temp_, Float *dt_,
	const Float hydro_dt, const Float previous_dt,
		const nnet::ptr_reaction_list &reactions, const func_type &construct_rates_BE, const func_eos &eos)
	{
	    const int i = blockIdx.x*blockDim.x + threadIdx.x;
	    if (i < n_particles) {
	    	Float *Mp        = (Float*)malloc((dimension + 1)*(dimension + 1)*sizeof(Float));
			Float *RHS       = (Float*)malloc(                (dimension + 1)*sizeof(Float));
			Float *DY_T      = (Float*)malloc(                (dimension + 1)*sizeof(Float));
			Float *Y_buffer  = (Float*)malloc(                      dimension*sizeof(Float));
			Float *rates     = (Float*)malloc(reactions.num_reactions*sizeof(Float));
			Float *drates_dT = (Float*)malloc(reactions.num_reactions*sizeof(Float));

		    if (rho_[i] > nnet::constants::dev_min_rho && temp_[i] > nnet::constants::dev_min_temp) {
				// compute drho/dt
				Float drho_dt = previous_rho_[i] <= 0 ? 0. : (rho_[i] - previous_rho_[i])/previous_dt;

				// solve
				nnet::solve_system_substep(dimension,
					Mp, RHS, DY_T, rates, drates_dT,
					reactions, construct_rates_BE, eos,
					&Y_[dimension*i], temp_[i], Y_buffer,
					rho_[i], drho_dt, hydro_dt, dt_[i]);
			}

			free(Mp);
			free(RHS);
			free(DY_T);
			free(Y_buffer);
			free(rates);
			free(drates_dT);
		}
	}

	// template<class func_type /*, class func_eos*/ /*, typename Float*/>
	__host__ void cudaComputeNuclearReactions(const int n_particles, const int dimension,
	Float *rho_, Float *previous_rho_, Float *Y_, Float *temp_, Float *dt_,
	const Float hydro_dt, const Float previous_dt,
		const nnet::ptr_reaction_list &reactions, const func_type &construct_rates_BE, const func_eos &eos)
	{
		int n_blocks                = CUDA_BLOCK_SIZE;
		int cuda_n_thread_per_block = (n_particles + n_blocks - 1) / n_blocks;

	    cudaKernelComputeNuclearReactions<<<n_blocks, cuda_n_thread_per_block>>>(n_particles, dimension,
			rho_, previous_rho_, Y_, temp_, dt_,
			hydro_dt, previous_dt,
			reactions, construct_rates_BE, eos);
	}
}
}