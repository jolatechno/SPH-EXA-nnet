#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "../../nuclear-net.hpp"

namespace sphexa {
namespace sphnnet {
	template<class func_type, class func_eos, typename Float>
	__global__ void cudaKernelComputeNuclearReactions(int dimension,
	Float *rho_, Float *previous_rho_, Float *Y_, Float *temp_, Float *dt_,
	const Float hydro_dt, const Float previous_dt,
		const nnet::ptr_reaction_list &reactions, const func_type &construct_rates_BE, const func_eos &eos)
	{
	    int i = threadIdx.x;

		Float Mp[(dimension + 1)*(dimension + 1)], RHS[dimension + 1], DY_T[dimension + 1], rates[reactions.num_reactions], drates_dT[reactions.num_reactions], Y_buffer[dimension];
	    
	    if (rho_[i] > nnet::constants::min_rho && temp_[i] > nnet::constants::min_temp) {
			// compute drho/dt
			Float drho_dt = previous_rho_[i] <= 0 ? 0. : (rho_[i] - previous_rho_[i])/previous_dt;

			// solve
			nnet::solve_system_substep(dimension,
				Mp, RHS, DY_T, rates, drates_dT,
				reactions, construct_rates_BE, eos,
				&Y_[dimension*i], temp_[i], Y_buffer,
				rho_[i], drho_dt, hydro_dt, dt_[i]);
		}
	}

	template<class func_type, class func_eos, typename Float>
	void cudaComputeNuclearReactions(int n_particles, int dimension,
	Float *rho_, Float *previous_rho_, Float *Y_, Float *temp_, Float *dt_,
	const Float hydro_dt, const Float previous_dt,
		const nnet::ptr_reaction_list &reactions, const func_type &construct_rates_BE, const func_eos &eos)
	{
	    cudaKernelComputeNuclearReactions<<<1, n_particles>>>(dimension,
			rho_, previous_rho_, Y_, temp_, dt_,
			hydro_dt, previous_dt,
			reactions, construct_rates_BE, eos);
	}
}
}